#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <algorithm>

// 使用原子操作的归约，必须在内核外把 out 初始化为 0
__global__ void
Reduction5_kernel(int* out, const int* in, size_t N) {
   const int tid = threadIdx.x;
   int partialSum = 0;
   for (size_t i = blockIdx.x * blockDim.x + tid;
        i < N;
        i += blockDim.x * gridDim.x) {
      partialSum += in[i];
   }
   atomicAdd(out, partialSum);
}

void Reduction5(int* answer, int* partial, const int* in, size_t N, int numBlocks, int numThreads) {
   hipMemset(answer, 0, sizeof(int));
   Reduction5_kernel<<<numBlocks, numThreads>>>(answer, in, N);
}

int main() {
   // malloc host memory
   size_t N = 1024;
   int *h_in = reinterpret_cast<int *>(std::malloc(N * sizeof(int)));
   std::generate(h_in, h_in + N, [n = 0] mutable
                 { return ++n; });

   size_t partialN = N;
   int numBlocks = 2;
   int numThreads = 4;

   // malloc device memory
   int *answer, *partial, *in;
   hipMalloc((void**)&answer, 1 * sizeof(int));
   hipMalloc((void**)&partial, partialN * sizeof(int));
   hipMalloc((void**)&in, N * sizeof(int));

   // transfer data from host to device
   hipMemcpy(in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

   // invoke the kernel
   for (int i = 0; i < 1000; ++i)
   {
      Reduction5(answer, partial, in, N, numBlocks, numThreads);
   }

   // transfer output from device to host
   int h_answer[1];
   hipMemcpy(h_answer, answer, 1 * sizeof(int), hipMemcpyDeviceToHost);
   printf("reduction sum: %d\n", h_answer[0]);
   return 0;
}
