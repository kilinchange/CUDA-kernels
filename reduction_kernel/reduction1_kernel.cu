#include <hip/hip_runtime.h>
#include <stdio.h>

// 两遍规约内核
__global__ void Reduction1_kernel(int* out, const int* in, size_t N) {
   extern __shared__ int sPartials[];
   int sum = 0;
   const int tid = threadIdx.x;
   // 对输入数组落入每个线程中的元素进行求和
   for (size_t i = blockIdx.x * blockDim.x + tid;
        i < N;
        i += blockDim.x * gridDim.x) {
      sum += in[i];
   }
   sPartials[tid] = sum;
   __syncthreads();

   // 针对共享内存中的值执行对数步长的归约操作
   for (int activeThreads = blockDim.x >> 1;
        activeThreads;
        activeThreads >>= 1) {
      if (tid < activeThreads) {
         sPartials[tid] += sPartials[tid + activeThreads];
      }
      __syncthreads();
   }

   // 线程块输出值写入全局内存
   if (tid == 0) {
      out[blockIdx.x] = sPartials[0];
   }
}

void Reduction1(int* answer, int* partial, const int* in, size_t N, int numBlocks, int numThreads) {
   unsigned int sharedSize = numThreads * sizeof(int);
   Reduction1_kernel<<<numBlocks, numThreads, sharedSize>>>(partial, in, N);
   Reduction1_kernel<<<1, numThreads, sharedSize>>>(answer, partial, numBlocks);
}

int main() {
   // malloc host memory
   int h_in[8] = {1, 2, 3, 4, 5, 6, 7, 8};

   // malloc device memory
   int *answer, *partial, *in;
   hipMalloc((void**)&answer, 1 * sizeof(int));
   hipMalloc((void**)&partial, 2 * sizeof(int));
   hipMalloc((void**)&in, 8 * sizeof(int));

   // transfer data from host to device
   hipMemcpy(in, h_in, 8 * sizeof(int), hipMemcpyHostToDevice);

   // invoke the kernel
   Reduction1(answer, partial, in, 8, 2, 4);

   // transfer output from device to host
   int h_answer[1];
   hipMemcpy(h_answer, answer, 1 * sizeof(int), hipMemcpyDeviceToHost);
   printf("reduction sum: %d\n", h_answer[0]);
   return 0;
}
